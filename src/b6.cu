#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "b6.cuh"
//////////////////////////////
//////////////////////////////

// __global__ void nb_1(const int* x, const float* y, float* z, int size, int n_feat, int n_classes) {
__global__ void nb_1(int n, float* z, int * x, float* y, int size, int n_feat, int n_classes) {         //n = 1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            }
        }
    }
}

 __global__ void nb_2(int n, float* y, float* x, int n_row_x, int n_col_x) {                            //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float curr_max = x[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            curr_max = fmaxf(curr_max, x[i * n_col_x + j]);
        }
        y[i] = curr_max;
    }
}

__global__ void nb_3(int n, float* z, float* x, float* y, int n_row_x, int n_col_x) {                   //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            sum += expf(x[i * n_col_x + j] - y[i]);
        }
        z[i] = logf(sum) + y[i];
    }
}

__global__ void nb_4(int n, float* x, float* y, int n_row_x, int n_col_x) {                             //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] = expf(x[i * n_col_x + j] - y[i]);
        }
    }
}

/*
__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

__global__ void rr_1_0(const int* x, float* y, float* z, int n_row_x, int n_col_x) {
    int warp_size = 32;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        // Compute mean and variance;
        float feature_mean = float(0);
        float sum_sq = float(0);
        for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n_row_x; i += blockDim.y * gridDim.y) {
            float x_tmp = x[j * n_row_x + i];
            feature_mean += x_tmp;
            sum_sq += x_tmp * x_tmp;
        }
        feature_mean = warp_reduce(feature_mean);  // Obtain the sum of values in the current warp;
        sum_sq = warp_reduce(sum_sq);              // Obtain the sum of values in the current warp;
        if (!(threadIdx.y % warp_size)) {
            atomicAdd(y + j, feature_mean);
            atomicAdd(z + j, sum_sq);
        }
    }
}

__global__ void rr_1_1(const int* x, float* y, const float* mean, const float* std, int n_row_x, int n_col_x) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        float mean_tmp = mean[j] / n_row_x;
        float std_tmp = sqrtf(std[j] / n_row_x - mean_tmp * mean_tmp);

        for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n_row_x; i += blockDim.y * gridDim.y) {
            y[j * n_row_x + i] = ((float)x[j * n_row_x + i] - mean_tmp) / std_tmp;
        }
    }
}
*/

__global__ void rr_1(int n, float* y, int* x, int n_row_x, int n_col_x) {                               //n=1
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        float feature_mean = 0;
        float sum_sq = 0;
        // Compute mean and variance;
        for (int i = 0; i < n_row_x; i++) {
            float x_tmp = x[j * n_row_x + i];
            feature_mean += x_tmp;
            sum_sq += x_tmp * x_tmp;
        }
        feature_mean /= n_row_x;
        float std = sqrtf(sum_sq / n_row_x - feature_mean * feature_mean);

        // Update values;
        for (int i = 0; i < n_row_x; i++) {
            y[j * n_row_x + i] = (x[j * n_row_x + i] - feature_mean) / std;
        }
    }
}

__global__ void rr_2(int n, float * z, float* x, float* y, int size, int n_feat, int n_classes) {       //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            }
        }
    }
}

 __global__ void rr_3(int n, float* x, float* y, int n_row_x, int n_col_x) {                            //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] += y[j];
        }
    }
}

 __global__ void softmax(int n, float* x, int n_row_x, int n_col_x) {                                   //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float row_exp_sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            row_exp_sum += expf(x[i * n_col_x + j]);
        }
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] = expf(x[i * n_col_x + j]) / row_exp_sum;
        }
    }
}

 __global__ void argmax(int n, int * z, float* x, float* y, int n_row_x, int n_col_x) {                 //n=1
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        int curr_best_index = 0;
        float curr_best = x[i * n_col_x] + y[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            float curr = x[i * n_col_x + j] + y[i * n_col_x + j];
            if (curr > curr_best) {
                curr_best = curr;
                curr_best_index = j;
            }
        }
        z[i] = curr_best_index;
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark6::alloc() {
    err = hipMallocManaged(&x, sizeof(int) * N * num_features);
    err = hipMallocManaged(&z, sizeof(float) * N * num_features);
    err = hipMallocManaged(&nb_feat_log_prob, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&nb_class_log_prior, sizeof(float) * num_classes);
    err = hipMallocManaged(&ridge_coeff, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&ridge_intercept, sizeof(float) * num_classes);
    err = hipMallocManaged(&nb_amax, sizeof(float) * N);
    err = hipMallocManaged(&nb_l, sizeof(float) * N);
    err = hipMallocManaged(&r1, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r2, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r, sizeof(int) * N);

    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
}

void Benchmark6::init() {
    for (int i = 0; i < num_classes; i++) {
        for (int j = 0; j < num_features; j++) {
            nb_feat_log_prob[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
            ridge_coeff[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
        }
        nb_class_log_prior[i] = (float)(rand()) / (float)(RAND_MAX);
        ridge_intercept[i] = (float)(rand()) / (float)(RAND_MAX);
    }
    int max_occurrence_of_ngram = 10;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_features; j++) {
            x[i * num_features + j] = rand() % max_occurrence_of_ngram;
        }
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
    }
}

void Benchmark6::reset() {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
        // r1_mean[i] = 0;
        // r1_std[i] = 0;
    }
}

void Benchmark6::execute_sync(int iter) {
    if (do_prefetch && pascalGpu) {
        hipMemPrefetchAsync(r1, sizeof(float) * N * num_classes, device_id, 0);
        hipDeviceSynchronize();
        hipMemPrefetchAsync(r2, sizeof(float) * N * num_classes, device_id, 0);
        hipDeviceSynchronize();
        hipMemPrefetchAsync(r, sizeof(int) * N, device_id, 0);
        hipDeviceSynchronize();
    }

    // rr_1<<<num_blocks, block_size_1d>>>(x, z, N, num_features);
    rr_1<<<num_blocks, block_size_1d>>>(1, z, x, N, num_features);
    // dim3 num_blocks_2d(8, 8);
    // dim3 block_size_1d_2d(1, 32);
    // rr_1_0<<<num_blocks_2d, block_size_1d_2d>>>(x, r1_mean, r1_std, N, num_features);
    // hipDeviceSynchronize();
    // rr_1_1<<<num_blocks_2d, block_size_1d_2d>>>(x, z, r1_mean, r1_std, N, num_features);
    hipDeviceSynchronize();

    // auto e1 = clock_type::now();
    // auto rr1time = chrono::duration_cast<chrono::microseconds>(e1 - start).count();
    // if (debug) std::cout << " rr1=" << (float) rr1time / 1000 << " ms" << std::endl;

    // nb_1<<<num_blocks, block_size_1d>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
    nb_1<<<num_blocks, block_size_1d>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);
    hipDeviceSynchronize();

    // rr_2<<<num_blocks, block_size_1d>>>(z, ridge_coeff, r2, N, num_features, num_classes);
    rr_2<<<num_blocks, block_size_1d>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);
    hipDeviceSynchronize();

    // nb_2<<<num_blocks, block_size_1d>>>(r1, nb_amax, N, num_classes);
    nb_2<<<num_blocks, block_size_1d>>>(1, nb_amax, r1, N, num_classes);
    hipDeviceSynchronize();

    // nb_3<<<num_blocks, block_size_1d>>>(r1, nb_amax, nb_l, N, num_classes);
    nb_3<<<num_blocks, block_size_1d>>>(1, nb_l, r1, nb_amax, N, num_classes);
    hipDeviceSynchronize();

    // rr_3<<<num_blocks, block_size_1d>>>(r2, ridge_intercept, N, num_classes);
    rr_3<<<num_blocks, block_size_1d>>>(1, r2, ridge_intercept, N, num_classes);
    hipDeviceSynchronize();

    // nb_4<<<num_blocks, block_size_1d>>>(r1, nb_l, N, num_classes);
    nb_4<<<num_blocks, block_size_1d>>>(1, r1, nb_l, N, num_classes);
    hipDeviceSynchronize();

    // softmax<<<num_blocks, block_size_1d>>>(r1, N, num_classes);
    softmax<<<num_blocks, block_size_1d>>>(1, r1, N, num_classes);
    hipDeviceSynchronize();

    // softmax<<<num_blocks, block_size_1d>>>(r2, N, num_classes);
    softmax<<<num_blocks, block_size_1d>>>(1, r2, N, num_classes);
    hipDeviceSynchronize();

    // argmax<<<num_blocks, block_size_1d>>>(r1, r2, r, N, num_classes);
    argmax<<<num_blocks, block_size_1d>>>(1, r, r1, r2, N, num_classes);
    hipDeviceSynchronize();
}

void FUNCb6(float * z, int * x, int N, int num_features, float * r1, float * nb_feat_log_prob, int num_classes, float * r2, float * ridge_coeff, 
            float * nb_amax, float * nb_l, float * ridge_intercept, int * r, int num_blocks, int block_size_1d)
{
    rr_1<<<num_blocks, block_size_1d>>>(1, z, x, N, num_features);

    nb_1<<<num_blocks, block_size_1d>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);

    rr_2<<<num_blocks, block_size_1d>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);

    nb_2<<<num_blocks, block_size_1d>>>(1, nb_amax, r1, N, num_classes);

    nb_3<<<num_blocks, block_size_1d>>>(1, nb_l, r1, nb_amax, N, num_classes);

    rr_3<<<num_blocks, block_size_1d>>>(1, r2, ridge_intercept, N, num_classes);

    nb_4<<<num_blocks, block_size_1d>>>(1, r1, nb_l, N, num_classes);

    softmax<<<num_blocks, block_size_1d>>>(1, r1, N, num_classes);

    softmax<<<num_blocks, block_size_1d>>>(1, r2, N, num_classes);

    argmax<<<num_blocks, block_size_1d>>>(1, r, r1, r2, N, num_classes);

}

void FUNCb6_prefetch(float * z, int * x, int N, int num_features, float * r1, float * nb_feat_log_prob, int num_classes, float * r2, float * ridge_coeff, 
            float * nb_amax, float * nb_l, float * ridge_intercept, int * r, int num_blocks, int block_size_1d,
            int prefetch_size1, int prefetch_size2, int device_id)
{
    hipMemPrefetchAsync(r1, prefetch_size1, device_id, 0);
    hipMemPrefetchAsync(r2, prefetch_size1, device_id, 0);
    hipMemPrefetchAsync(r, prefetch_size2, device_id, 0);

    rr_1<<<num_blocks, block_size_1d>>>(1, z, x, N, num_features);

    nb_1<<<num_blocks, block_size_1d>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);

    rr_2<<<num_blocks, block_size_1d>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);

    nb_2<<<num_blocks, block_size_1d>>>(1, nb_amax, r1, N, num_classes);

    nb_3<<<num_blocks, block_size_1d>>>(1, nb_l, r1, nb_amax, N, num_classes);

    rr_3<<<num_blocks, block_size_1d>>>(1, r2, ridge_intercept, N, num_classes);

    nb_4<<<num_blocks, block_size_1d>>>(1, r1, nb_l, N, num_classes);

    softmax<<<num_blocks, block_size_1d>>>(1, r1, N, num_classes);

    softmax<<<num_blocks, block_size_1d>>>(1, r2, N, num_classes);

    argmax<<<num_blocks, block_size_1d>>>(1, r, r1, r2, N, num_classes);

}

void Benchmark6::execute_AssOfKFC(int iter)
{
    // if (do_prefetch && pascalGpu) {
    //     hipMemPrefetchAsync(r1, sizeof(float) * N * num_classes, device_id, 0);
    //     hipMemPrefetchAsync(r2, sizeof(float) * N * num_classes, device_id, 0);
    //     hipMemPrefetchAsync(r, sizeof(int) * N, device_id, 0);
    //     hipDeviceSynchronize();
    // }
    if(!do_prefetch || !pascalGpu) FUNCb6(z, x, N, num_features, r1, nb_feat_log_prob, num_classes, r2, ridge_coeff, nb_amax, nb_l, ridge_intercept, r, num_blocks, block_size_1d);
    if(do_prefetch && pascalGpu)  FUNCb6_prefetch(z, x, N, num_features, r1, nb_feat_log_prob, num_classes, r2, ridge_coeff, nb_amax, nb_l, ridge_intercept, r, num_blocks, block_size_1d, 
                                                    sizeof(float) * N * num_classes, sizeof(int) * N, device_id);
    err = hipGetLastError();
    if (debug && err) std::cout << err << std::endl;
}

void Benchmark6::execute_async(int iter) {
    if (!pascalGpu || stream_attach) {
        hipStreamAttachMemAsync(s1, z, 0);
        // hipStreamAttachMemAsync(s1, r1_mean, 0);
        // hipStreamAttachMemAsync(s1, r1_std, 0);
        hipStreamAttachMemAsync(s2, nb_feat_log_prob, 0);
        hipStreamAttachMemAsync(s2, r1, 0);
        hipStreamAttachMemAsync(s1, ridge_coeff, 0);
        hipStreamAttachMemAsync(s1, r2, 0);
        hipStreamAttachMemAsync(s2, nb_amax, 0);
        hipStreamAttachMemAsync(s2, nb_l, 0);
        hipStreamAttachMemAsync(s1, ridge_intercept, 0);
    }
    if (do_prefetch && pascalGpu) {
        hipMemPrefetchAsync(r1, sizeof(float) * N * num_classes, device_id, s2);
        hipMemPrefetchAsync(r2, sizeof(float) * N * num_classes, device_id, s1);
        hipMemPrefetchAsync(r, sizeof(int) * N, device_id, s1);
    }

    // rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
    rr_1<<<num_blocks, block_size_1d, 0, s1>>>(1, z, x, N, num_features);

    // dim3 num_blocks_2d(8, 8);
    // dim3 block_size_1d_2d(8, 8);
    // rr_1_0<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, r1_mean, r1_std, N, num_features);
    // rr_1_1<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, z, r1_mean, r1_std, N, num_features);

    // nb_1<<<num_blocks, block_size_1d, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
    nb_1<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);

    // rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);
    rr_2<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);

    // nb_2<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, N, num_classes);
    nb_2<<<num_blocks, block_size_1d, 0, s2>>>(1, nb_amax, r1, N, num_classes);

    // nb_3<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);
    nb_3<<<num_blocks, block_size_1d, 0, s2>>>(1, nb_l, r1, nb_amax, N, num_classes);

    // rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);
    rr_3<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, ridge_intercept, N, num_classes);

    // nb_4<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_l, N, num_classes);
    nb_4<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, nb_l, N, num_classes);

    // softmax<<<num_blocks, block_size_1d, 0, s2>>>(r1, N, num_classes);
    softmax<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, N, num_classes);

    // softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes);
    softmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, N, num_classes);

    // Stream 1 waits stream 2;
    hipEvent_t e1;
    hipEventCreate(&e1);
    hipEventRecord(e1, s2);
    hipStreamWaitEvent(s1, e1, 0);

    // argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);
    argmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r, r1, r2, N, num_classes);
    hipDeviceSynchronize();
}

void Benchmark6::execute_cudagraph(int iter) {
    if (iter == 0) {
        hipEvent_t ef;
        hipEventCreate(&ef);
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);
        hipEventRecord(ef, s1);
        hipStreamWaitEvent(s2, ef, 0);

        // rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
        rr_1<<<num_blocks, block_size_1d, 0, s1>>>(1, z, x, N, num_features);

        // dim3 num_blocks_2d(8, 8);
        // dim3 block_size_1d_2d(8, 8);
        // rr_1_0<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, r1_mean, r1_std, N, num_features);
        // rr_1_1<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, z, r1_mean, r1_std, N, num_features);

        // nb_1<<<num_blocks, block_size_1d, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
        nb_1<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);

        // rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);
        rr_2<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);

        // nb_2<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, N, num_classes);
        nb_2<<<num_blocks, block_size_1d, 0, s2>>>(1, nb_amax, r1, N, num_classes);

        // nb_3<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);
        nb_3<<<num_blocks, block_size_1d, 0, s2>>>(1, nb_l, r1, nb_amax, N, num_classes);

        // rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);
        rr_3<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, ridge_intercept, N, num_classes);

        // nb_4<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_l, N, num_classes);
        nb_4<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, nb_l, N, num_classes);

        // softmax<<<num_blocks, block_size_1d, 0, s2>>>(r1, N, num_classes);
        softmax<<<num_blocks, block_size_1d, 0, s2>>>(1, r1, N, num_classes);

        // softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes);
        softmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, N, num_classes);

        // Stream 1 waits stream 2;
        hipEvent_t e1;
        hipEventCreate(&e1);
        hipEventRecord(e1, s2);
        hipStreamWaitEvent(s1, e1, 0);

        // argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);
        argmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r, r1, r2, N, num_classes);

        hipStreamEndCapture(s1, &graph);
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

void Benchmark6::execute_cudagraph_manual(int iter) {
    if (iter == 0) {
        hipGraphCreate(&graph, 0);
        // void* kernel_1_args[4] = {(void*)&x, (void*)&z, &N, &num_features};
        // void* kernel_2_args[6] = {(void*)&x, (void*)&nb_feat_log_prob, (void*)&r1, &N, &num_features, &num_classes};
        // void* kernel_3_args[6] = {(void*)&z, (void*)&ridge_coeff, (void*)&r2, &N, &num_features, &num_classes};
        // void* kernel_4_args[4] = {(void*)&r1, (void*)&nb_amax, &N, &num_classes};
        // void* kernel_5_args[5] = {(void*)&r1, (void*)&nb_amax, (void*)&nb_l, &N, &num_classes};
        // void* kernel_6_args[4] = {(void*)&r2, (void*)&ridge_intercept, &N, &num_classes};
        // void* kernel_7_args[4] = {(void*)&r1, (void*)&nb_l, &N, &num_classes};
        // void* kernel_8_args[3] = {(void*)&r1, &N, &num_classes};
        // void* kernel_9_args[3] = {(void*)&r2, &N, &num_classes};
        // void* kernel_10_args[5] = {(void*)&r1, (void*)&r2, (void*)&r, &N, &num_classes};

        int output_n = 1;

        void* kernel_1_args[5] = {&output_n, (void*)&z, (void*)&x, &N, &num_features};
        void* kernel_2_args[7] = {&output_n, (void*)&r1, (void*)&x, (void*)&nb_feat_log_prob, &N, &num_features, &num_classes};
        void* kernel_3_args[7] = {&output_n, (void*)&r2, (void*)&z, (void*)&ridge_coeff, &N, &num_features, &num_classes};
        void* kernel_4_args[5] = {&output_n, (void*)&nb_amax, (void*)&r1, &N, &num_classes};
        void* kernel_5_args[6] = {&output_n, (void*)&nb_l, (void*)&r1, (void*)&nb_amax, &N, &num_classes};
        void* kernel_6_args[5] = {&output_n, (void*)&r2, (void*)&ridge_intercept, &N, &num_classes};
        void* kernel_7_args[5] = {&output_n, (void*)&r1, (void*)&nb_l, &N, &num_classes};
        void* kernel_8_args[4] = {&output_n, (void*)&r1, &N, &num_classes};
        void* kernel_9_args[4] = {&output_n, (void*)&r2, &N, &num_classes};
        void* kernel_10_args[6] = {&output_n, (void*)&r, (void*)&r1, (void*)&r2, &N, &num_classes};

        dim3 tb(block_size_1d);
        dim3 bs(num_blocks);

        add_node(kernel_1_args, kernel_1_params, (void*)rr_1, bs, tb, graph, &kernel_1, nodeDependencies);
        add_node(kernel_2_args, kernel_2_params, (void*)nb_1, bs, tb, graph, &kernel_2, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_1);
        add_node(kernel_3_args, kernel_3_params, (void*)rr_2, bs, tb, graph, &kernel_3, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_2);
        add_node(kernel_4_args, kernel_4_params, (void*)nb_2, bs, tb, graph, &kernel_4, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_4);
        add_node(kernel_5_args, kernel_5_params, (void*)nb_3, bs, tb, graph, &kernel_5, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_3);
        add_node(kernel_6_args, kernel_6_params, (void*)rr_3, bs, tb, graph, &kernel_6, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_5);
        add_node(kernel_7_args, kernel_7_params, (void*)nb_4, bs, tb, graph, &kernel_7, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_7);
        add_node(kernel_8_args, kernel_8_params, (void*)softmax, bs, tb, graph, &kernel_8, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_6);
        add_node(kernel_9_args, kernel_9_params, (void*)softmax, bs, tb, graph, &kernel_9, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_8);
        nodeDependencies.push_back(kernel_9);
        add_node(kernel_10_args, kernel_10_params, (void*)argmax, bs, tb, graph, &kernel_10, nodeDependencies);

        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

void Benchmark6::execute_cudagraph_single(int iter) {
    if (iter == 0) {
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);

        // rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
        rr_1<<<num_blocks, block_size_1d, 0, s1>>>(1, z, x, N, num_features);

        // dim3 num_blocks_2d(8, 8);
        // dim3 block_size_1d_2d(8, 8);
        // rr_1_0<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, r1_mean, r1_std, N, num_features);
        // rr_1_1<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, z, r1_mean, r1_std, N, num_features);

        // nb_1<<<num_blocks, block_size_1d, 0, s1>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
        nb_1<<<num_blocks, block_size_1d, 0, s1>>>(1, r1, x, nb_feat_log_prob, N, num_features, num_classes);

        // rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);
        rr_2<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, z, ridge_coeff, N, num_features, num_classes);

        // nb_2<<<num_blocks, block_size_1d, 0, s1>>>(r1, nb_amax, N, num_classes);
        nb_2<<<num_blocks, block_size_1d, 0, s1>>>(1, nb_amax, r1, N, num_classes);

        // nb_3<<<num_blocks, block_size_1d, 0, s1>>>(r1, nb_amax, nb_l, N, num_classes);
        nb_3<<<num_blocks, block_size_1d, 0, s1>>>(1, nb_l, r1, nb_amax, N, num_classes);

        // rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);
        rr_3<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, ridge_intercept, N, num_classes);
        
        // nb_4<<<num_blocks, block_size_1d, 0, s1>>>(r1, nb_l, N, num_classes);
        nb_4<<<num_blocks, block_size_1d, 0, s1>>>(1, r1, nb_l, N, num_classes);

        // softmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, N, num_classes);
        softmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r1, N, num_classes);

        // softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes);
        softmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r2, N, num_classes);

        // argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);
        argmax<<<num_blocks, block_size_1d, 0, s1>>>(1, r, r1, r2, N, num_classes);

        hipStreamEndCapture(s1, &graph);
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

std::string Benchmark6::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(r[0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < 10; j++) {
            res += std::to_string(r[j]) + ", ";
        }
        return res + ", ...]";
    }
}

Benchmark6::~Benchmark6()
{
    // err = hipMallocManaged(&x, sizeof(int) * N * num_features);
    // err = hipMallocManaged(&z, sizeof(float) * N * num_features);
    // err = hipMallocManaged(&nb_feat_log_prob, sizeof(float) * num_classes * num_features);
    // err = hipMallocManaged(&nb_class_log_prior, sizeof(float) * num_classes);
    // err = hipMallocManaged(&ridge_coeff, sizeof(float) * num_classes * num_features);
    // err = hipMallocManaged(&ridge_intercept, sizeof(float) * num_classes);
    // err = hipMallocManaged(&nb_amax, sizeof(float) * N);
    // err = hipMallocManaged(&nb_l, sizeof(float) * N);
    // err = hipMallocManaged(&r1, sizeof(float) * N * num_classes);
    // err = hipMallocManaged(&r2, sizeof(float) * N * num_classes);
    // err = hipMallocManaged(&r, sizeof(int) * N);

    // err = hipStreamCreate(&s1);
    // err = hipStreamCreate(&s2);

    hipFree(x);
    hipFree(z);
    hipFree(nb_feat_log_prob);
    hipFree(nb_class_log_prior);
    hipFree(ridge_coeff);
    hipFree(ridge_intercept);
    hipFree(nb_amax);
    hipFree(nb_l);
    hipFree(r1);
    hipFree(r2);
    hipFree(r);

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
}